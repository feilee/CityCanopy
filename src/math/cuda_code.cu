#include "hip/hip_runtime.h"
#include "cuda_code.cuh"
#include "../../third_party/others/OpenGlCudaHelper.h"
#include <algorithm>


__global__ void 
gol_step
(
    const ubyte_ptr m_data,
    ubyte_ptr       m_data_out,
    const unsigned  matrix_height,
    const unsigned  matrix_width
)
{
    unsigned n_size = matrix_height * matrix_width;

    for (unsigned cell_id = blockIdx.x * blockDim.x + threadIdx.x;
        cell_id < n_size;
        cell_id += blockDim.x * gridDim.x) 
    {

        uint x1 = cell_id % matrix_width;
        uint y1 = cell_id - x1;     // position 0 to matrix y1 as array

        uint x0 = (x1 + matrix_width - 1) % matrix_width;   // left
        uint x2 = (x1 + 1) % matrix_width;                  // right

        uint y0 = (y1 + n_size - matrix_width) % n_size;    // up
        uint y2 = (y1 + matrix_width) % n_size;             // down

        // Count alive cells.
        uint alive_cells = 
                  m_data[x0 + y0] + m_data[x1 + y0] + m_data[x2 + y0]
                + m_data[x0 + y1] +        0        + m_data[x2 + y1]
                + m_data[x0 + y2] + m_data[x1 + y2] + m_data[x2 + y2];

        m_data_out[x1 + y1] =
            (
                alive_cells == 3 || 
                alive_cells == 6 || 
                (alive_cells == 2 && m_data[x1 + y1])
            ) ? 1 : 0;
    }
}

bool gol_iterate
(
    ubyte_ptr&  m_data,
    ubyte_ptr&  m_data_out,
    const size_t matrix_height,
    const size_t matrix_width,
    const size_t iterations,
    const ushort threads
) 
{
    //if ((matrix_height * matrix_width) % threads != 0)
    //    return false;

    size_t required_blocks = (matrix_height * matrix_width) / threads;
    ushort blocks = (ushort) std::min((size_t)32768, required_blocks);

    for (size_t i = 0; i < iterations; ++i) 
    {
        gol_step<<<blocks, threads>>>
            (m_data, m_data_out, uint(matrix_height), uint(matrix_width));
        std::swap(m_data, m_data_out);
    }
    mf::checkCudaErrors(hipDeviceSynchronize());

    return true;
}
